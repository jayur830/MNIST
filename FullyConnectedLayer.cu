#include "FullyConnectedLayer.cuh"
#include "hiprand/hiprand_kernel.h"
#include ""

FC::FC() {}

FC::FC(hipblasHandle_t* _blasHandle, hipdnnHandle_t* _handle, const int _batchSize, const int _inputSize, const int _outputSize) :
	blasHandle(_blasHandle), handle(_handle), batchSize(_batchSize), inputSize(_inputSize), outputSize(_outputSize) {}

FullyConnectedLayer::FullyConnectedLayer() : batchSize(1), inputSize(1), outputSize(1) {}

FullyConnectedLayer::FullyConnectedLayer(hipblasHandle_t* _blasHandle, hipdnnHandle_t* _handle, const int _batchSize, const int _inputSize, const int _outputSize) :
	batchSize(_batchSize), inputSize(_inputSize), outputSize(_outputSize) {	
	initialize(_blasHandle, _handle);
}

FullyConnectedLayer::FullyConnectedLayer(FC fc) :
	batchSize(fc.batchSize), inputSize(fc.inputSize), outputSize(fc.outputSize) {
	initialize(fc.blasHandle, fc.handle);
}

FullyConnectedLayer::~FullyConnectedLayer() {
	checkCUDNN(hipdnnDestroyTensorDescriptor(this->outputDesc));

	checkCUDA(hipFree(this->weights));
	checkCUDA(hipFree(this->one_vec));
	checkCUDA(hipFree(this->bias));
	checkCUDA(hipFree(this->output));
	checkCUDA(hipFree(this->g_weights));
	checkCUDA(hipFree(this->g_bias));
	checkCUDA(hipFree(this->grad));
}

void FullyConnectedLayer::forwardProp(float* input) {
	checkCUBLAS(hipblasSgemm(*this->blasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, this->outputSize, this->batchSize, this->inputSize,
		& this->alpha, this->weights, this->inputSize, this->input = input, this->inputSize, & this->beta, this->output, this->outputSize));
	checkCUBLAS(hipblasSgemm(*this->blasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, this->outputSize, this->batchSize,
		1, & this->alpha, this->bias, this->outputSize, this->one_vec, 1, & this->alpha, this->output, this->outputSize));
}

void FullyConnectedLayer::backProp(float* grad) {
	checkCUBLAS(hipblasSgemm(*this->blasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, this->inputSize, this->outputSize, this->batchSize,
		&this->alpha, this->input, this->inputSize, grad, this->outputSize, &this->beta, this->g_weights, this->inputSize));
	checkCUBLAS(hipblasSgemv(*this->blasHandle, HIPBLAS_OP_N, this->outputSize, this->batchSize,
		&this->alpha, grad, this->outputSize, this->one_vec, 1, &this->beta, this->g_bias, 1));
	checkCUBLAS(hipblasSgemm(*this->blasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, this->inputSize, this->batchSize, this->outputSize,
		&this->alpha, this->weights, this->inputSize, grad, this->outputSize, &this->beta, this->grad, this->inputSize));
}

void FullyConnectedLayer::update(const float learningRate) {
	float learn(learningRate);
	checkCUBLAS(hipblasSaxpy(*this->blasHandle, static_cast<int>(this->inputSize * this->outputSize), &learn, this->g_weights, 1, this->weights, 1));
	checkCUBLAS(hipblasSaxpy(*this->blasHandle, static_cast<int>(this->inputSize * this->outputSize), &learn, this->g_bias, 1, this->bias, 1));
}

float* FullyConnectedLayer::get() {
	return this->output;
}

float* FullyConnectedLayer::gradient() {
	return this->grad;
}

hipdnnTensorDescriptor_t FullyConnectedLayer::descriptor() {
	return this->outputDesc;
}

void FullyConnectedLayer::initialize(hipblasHandle_t* _blasHandle, hipdnnHandle_t* _handle) {
	this->blasHandle = _blasHandle;
	this->handle = _handle;

	checkCUDNN(hipdnnCreateTensorDescriptor(&this->outputDesc));
	checkCUDNN(hipdnnSetTensor4dDescriptor(this->outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, this->batchSize, this->outputSize, 1, 1));

	checkCUDA(hipMalloc(&this->weights, sizeof(float) * this->inputSize * this->outputSize));
	checkCUDA(hipMalloc(&this->bias, sizeof(float) * this->outputSize));
	checkCUDA(hipMalloc(&this->one_vec, sizeof(float) * this->batchSize));
	checkCUDA(hipMalloc(&this->output, sizeof(float) * this->batchSize * this->outputSize));
	checkCUDA(hipMalloc(&this->g_output, sizeof(float) * this->batchSize * this->outputSize));
	checkCUDA(hipMalloc(&this->g_weights, sizeof(float) * this->inputSize * this->outputSize));
	checkCUDA(hipMalloc(&this->g_bias, sizeof(float) * this->outputSize));
	checkCUDA(hipMalloc(&this->grad, sizeof(float) * this->batchSize * this->inputSize));

	float* w(new float[this->inputSize * this->outputSize]), * b(new float[this->outputSize]);
	xavier(this->inputSize * this->outputSize, w, this->inputSize * this->outputSize, b, this->outputSize);
	checkCUDA(hipMemcpy(this->weights, w, sizeof(float) * this->inputSize * this->outputSize, hipMemcpyHostToDevice));
	checkCUDA(hipMemcpy(this->bias, b, sizeof(float) * this->outputSize, hipMemcpyHostToDevice));
	delete[] w, b;
	w = nullptr; b = nullptr;
}