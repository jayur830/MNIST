#include "ActivationLayer.cuh"

Act::Act() {}

Act::Act(hipblasHandle_t* _blasHandle, hipdnnHandle_t* _handle, const int _batchSize, const int _size,
	hipdnnActivationMode_t _mode) :
	blasHandle(_blasHandle), handle(_handle), batchSize(_batchSize), inputSize(_size), outputSize(_size), mode(_mode) {}

ActivationLayer::ActivationLayer() :
	batchSize(1), inputSize(1), outputSize(1), mode(HIPDNN_ACTIVATION_RELU) {}

ActivationLayer::ActivationLayer(hipblasHandle_t* _blasHandle, hipdnnHandle_t* _handle, const int _batchSize, const int _size, hipdnnActivationMode_t _mode) :
	batchSize(_batchSize), inputSize(_size), outputSize(_size), mode(_mode) {
	initialize(_blasHandle, _handle);
}

ActivationLayer::ActivationLayer(Act act) :
	batchSize(act.batchSize), inputSize(act.inputSize), outputSize(act.outputSize), mode(act.mode) {
	initialize(act.blasHandle, act.handle);
}

ActivationLayer::~ActivationLayer() {
	checkCUDNN(hipdnnDestroyTensorDescriptor(this->outputDesc));
	checkCUDNN(hipdnnDestroyActivationDescriptor(this->actDesc));

	checkCUDA(hipFree(this->output));
	checkCUDA(hipFree(this->grad));
}

void ActivationLayer::forwardProp(float* input) {
	checkCUDNN(hipdnnActivationForward(*this->handle, this->actDesc, &this->alpha, this->outputDesc, this->input = input, &this->beta, this->outputDesc, this->output));
}

void ActivationLayer::backProp(float* grad) {
	checkCUDNN(hipdnnActivationBackward(*this->handle, this->actDesc, &this->alpha, this->outputDesc,
		this->output, this->outputDesc, grad, this->outputDesc, this->input, &this->beta, this->outputDesc, this->grad));
}

float* ActivationLayer::get() {
	return this->output;
}

float* ActivationLayer::gradient() {
	return this->grad;
}

void ActivationLayer::initialize(hipblasHandle_t* _blasHandle, hipdnnHandle_t* _handle) {
	this->blasHandle = _blasHandle;
	this->handle = _handle;
	checkCUDNN(hipdnnCreateTensorDescriptor(&this->outputDesc));
	checkCUDNN(hipdnnCreateActivationDescriptor(&this->actDesc));
	checkCUDNN(hipdnnSetTensor4dDescriptor(this->outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, this->batchSize, this->outputSize, 1, 1));
	checkCUDNN(hipdnnSetActivationDescriptor(this->actDesc, this->mode, HIPDNN_PROPAGATE_NAN, 1.0));
	checkCUDA(hipMalloc(&this->output, sizeof(float) * this->batchSize * this->outputSize));
	checkCUDA(hipMalloc(&this->grad, sizeof(float) * this->batchSize * this->inputSize));
}