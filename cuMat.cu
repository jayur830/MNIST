#include "hip/hip_runtime.h"
#include ""
//#include "hiprand/hiprand_kernel.h"
#include "cuMat.cuh"
#include <iostream>

cuMat::cuMat() {}

cuMat::cuMat(float* data, int size) : _size(size) {
	hipMalloc(&this->mat, sizeof(float) * size);
	hipMemcpy(this->mat, data, sizeof(float) * size, hipMemcpyHostToDevice);
}

cuMat::cuMat::cuMat(float* data, int height, int width) : _size(height * width) {
	hipMalloc(&this->mat, sizeof(float) * height * width);
	hipMemcpy(this->mat, data, sizeof(float) * height * width, hipMemcpyHostToDevice);
}

cuMat::cuMat(float** data, int height, int width) : _size(height* width) {
	hipMalloc(&this->mat, sizeof(float) * height * width);
	hipMemcpy(this->mat, data, sizeof(float) * height * width, hipMemcpyHostToDevice);
}

cuMat::cuMat(std::vector<float> data) : _size(data.size()) {
	hipMalloc(&this->mat, sizeof(float) * data.size());
	hipMemcpy(this->mat, data.data(), sizeof(float) * data.size(), hipMemcpyHostToDevice);
}

cuMat::cuMat(std::vector<float> data, int height, int width) : _size(height * width) {
	hipMalloc(&this->mat, sizeof(float) * height * width);
	hipMemcpy(this->mat, data.data(), sizeof(float) * height * width, hipMemcpyHostToDevice);
	//float* _data(new float[this->_size]);
	//hipMemcpy(_data, this->mat, sizeof(float) * this->_size, hipMemcpyDeviceToHost);
	//for (int i(0); i < height; ++i) {
	//	for (int j(0); j < width; ++j)
	//		std::cout << _data[i * width + j] << " ";
	//	std::cout << std::endl;
	//}
	//delete[] _data;
	//_data = nullptr;
}

cuMat::cuMat(std::vector<std::vector<float>> data) : _size(data.size() * data[0].size()) {
	hipMalloc(&this->mat, sizeof(float) * data.size() * data[0].size());
	float* temp(new float[data.size() * data[0].size()]);
	for (int i(0); i < data.size(); ++i)
		for (int j(0); j < data[i].size(); ++j)
			temp[i * data[i].size() + j] = data[i][j];
	hipMemcpy(this->mat, temp, sizeof(float) * data.size() * data[0].size(), hipMemcpyHostToDevice);
	delete[] temp;
	temp = nullptr;
}

cuMat::~cuMat() {
	hipFree(this->mat);
}

float* cuMat::getMatrix(bool gpu) {
	if (gpu) return this->mat;
	else {
		float* _data(new float[this->_size]);
		hipMemcpy(_data, this->mat, sizeof(float) * this->_size, hipMemcpyDeviceToHost);
		return _data;
	}
}

size_t cuMat::size() {
	return this->_size;
}