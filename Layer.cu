#include "hip/hip_runtime.h"
#include "Layer.cuh"
#include ""
#include <random>

const char* cublasGetErrorString(hipblasStatus_t status) {
	switch (status) {
	case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
	case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
	case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
	case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
	case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
	case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
	case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
	case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}
	return "unknown error";
}

Layer::Layer() {
	alpha = 1.0;
	beta = 0;
}

Layer::~Layer() {}

void Layer::xavier(float value, float* weights, int weightSize, float* bias, int biasSize) {
	std::random_device rd;
	std::mt19937 random(rd());
	float w(sqrt(3.0 / value));
	std::uniform_real_distribution<> dist(-w, w);
	for (int i(0); i < weightSize; ++i)
		weights[i] = static_cast<float>(dist(random));
	for (int i(0); i < biasSize; ++i)
		bias[i] = static_cast<float>(dist(random));
}

__global__ void seed_kernel(hiprandState* state, int seed) {
	int index(blockIdx.x * blockDim.x + threadIdx.x);
	hiprand_init(seed, index, 0, &state[index]);
}

__global__ void init_kernel(hiprandState* state, float* arr) {
	int index(blockIdx.x * blockDim.x + threadIdx.x);
	arr[index] = hiprand_uniform(&state[index]);
}

__global__ void init_one_vec_kernel(float* arr) {
	arr[threadIdx.x] = 1.0;
}